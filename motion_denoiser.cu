#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include "denoiser_cuda.h"

__global__ void simpleDenoiseKernel(
    const uint8_t* cur, const uint8_t* prev, const uint8_t* next,
    uint8_t* dst, int w, int h, int stride,
    int strength, float temporal)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    int idx = y * stride + x;
    float val = cur[idx];

    // ���ԕ�������
    if (prev && next) {
        val = (1.0f - temporal) * val
            + temporal * 0.5f * (prev[idx] + next[idx]);
    }
    else if (prev) {
        val = (1.0f - temporal) * val + temporal * prev[idx];
    }
    else if (next) {
        val = (1.0f - temporal) * val + temporal * next[idx];
    }

    // ��ԕ��� 3x3 �ڂ���
    float sum = 0.0f;
    int count = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int xx = min(max(x + dx, 0), w - 1);
            int yy = min(max(y + dy, 0), h - 1);
            sum += cur[yy * stride + xx];
            count++;
        }
    }
    float spatial = sum / count;

    // ���ԂƋ�Ԃ̃u�����h
    float blended = (1.0f - strength / 100.0f) * val
        + (strength / 100.0f) * spatial;

    dst[idx] = (uint8_t)min(max(int(blended + 0.5f), 0), 255);
}

void run_simple_denoise(
    const uint8_t* cur,
    const uint8_t* prev,
    const uint8_t* next,
    uint8_t* dst,
    int w, int h, int stride,
    int strength, float temporal)
{
    size_t frame_size = stride * h;
    uint8_t* d_cur, * d_prev = nullptr, * d_next = nullptr, * d_dst;

    hipMalloc(&d_cur, frame_size);
    hipMalloc(&d_dst, frame_size);
    if (prev) hipMalloc(&d_prev, frame_size);
    if (next) hipMalloc(&d_next, frame_size);

    hipMemcpy(d_cur, cur, frame_size, hipMemcpyHostToDevice);
    if (prev) hipMemcpy(d_prev, prev, frame_size, hipMemcpyHostToDevice);
    if (next) hipMemcpy(d_next, next, frame_size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((w + 15) / 16, (h + 15) / 16);
    simpleDenoiseKernel << <blocks, threads >> > (
        d_cur, d_prev, d_next,
        d_dst, w, h, stride, strength, temporal);

    hipMemcpy(dst, d_dst, frame_size, hipMemcpyDeviceToHost);

    hipFree(d_cur);
    hipFree(d_dst);
    if (prev) hipFree(d_prev);
    if (next) hipFree(d_next);
}
